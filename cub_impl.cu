#include "hip/hip_runtime.h"
/*
 * cub_impl.cu
 * Copyright (C) 2016- CloudBrain <byzhang@>
 */

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include "cub_impl.h"
#include "distance.h"

#include <hipcub/hipcub.hpp>
#include <glog/logging.h>

using namespace hipcub;
using namespace std;

static const uint32_t num_data_per_block = 1024;
static hipcub::CachingDeviceAllocator allocator_;

kNN::impl::~impl() {
  if (tex_) {
    hipDestroyTextureObject(tex_);
  }
  if (data_) {
    hipFreeArray(data_);
  }
}

kNN::impl::impl(const std::vector<uint32_t>& data, uint32_t num_data, uint32_t num_dim)
  : num_data_(num_data), num_dim_(num_dim)
  , tex_height_(num_data / num_data_per_block + ((num_data % num_data_per_block)? 1: 0)) {
  CHECK(data.size() == num_data * num_dim) << "size mismatch:"
      << "data = " << data.size()
      << ", num_data * num_dim = " << num_data * num_dim;

  // allocate array
  auto channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);
  auto error = hipMallocArray(&data_, &channelDesc, num_dim * num_data_per_block, tex_height_);
  CHECK(error == hipSuccess) << "error " << error << " when allocating data_ "
                              << "num_dim:" << num_dim_ << "; num_data:" << num_data_ << "; num_data_per_block:" << num_data_per_block
                              << "; tex_height:" << tex_height_;

  // memcpy array
  error = hipMemcpyToArray(data_, 0, 0, data.data(), num_data_ * num_dim_ * sizeof(uint32_t), hipMemcpyHostToDevice);
  CHECK(error == hipSuccess) << "error " << error << " when copying to data_"
                              << "num_dim:" << num_dim_ << "; num_data:" << num_data_ << "; num_data_per_block:" << num_data_per_block
                              << "; tex_height:" << tex_height_;

  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = data_;

  // Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  // Create texture object
  error = hipCreateTextureObject(&tex_, &resDesc, &texDesc, NULL);
  CHECK(error == hipSuccess) << "error " << error << " when copying to data_"
                              << "num_dim:" << num_dim_ << "; num_data:" << num_data_ << "; num_data_per_block:" << num_data_per_block
                              << "; tex_height:" << tex_height_;
}

#define CubDebugReturn(e, r) if (hipcub::Debug((e), __FILE__, __LINE__)) { return r; }

std::vector<uint32_t> kNN::impl::search(const std::vector<uint32_t>& query, uint32_t top_k) {
  std::vector<uint32_t> indexes{};
  if (query.size() != num_dim_) {
    LOG_EVERY_N(ERROR, 10000) << "size mismatch:"
                              << "query = " << query.size()
                              << ", num_dim = " << num_dim_;
    return indexes;
  }

  // TODO: thread local
  uint32_t* query_device = nullptr;
  auto error = allocator_.DeviceAllocate((void**)&query_device, sizeof(uint32_t) * num_dim_);
  if (error != hipSuccess) {
    LOG_EVERY_N(ERROR, 1000) << "error " << error << " when aollcating query_device:" << num_dim_;
    return indexes;
  }

  CubDebugReturn(hipMemcpy(query_device, query.data(), sizeof(uint32_t) * num_dim_, hipMemcpyHostToDevice), indexes);

  // TODO: thread local
  DoubleBuffer<uint16_t> d_keys;
  DoubleBuffer<uint32_t> d_values;
  CubDebugReturn(allocator_.DeviceAllocate((void**)&d_keys.d_buffers[0], sizeof(uint16_t) * num_data_), indexes);
  CubDebugReturn(allocator_.DeviceAllocate((void**)&d_keys.d_buffers[1], sizeof(uint16_t) * num_data_), indexes);
  CubDebugReturn(allocator_.DeviceAllocate((void**)&d_values.d_buffers[0], sizeof(uint32_t) * num_data_), indexes);
  CubDebugReturn(allocator_.DeviceAllocate((void**)&d_values.d_buffers[1], sizeof(uint32_t) * num_data_), indexes);

  auto* keys = d_keys.d_buffers[0];
  auto* values = d_values.d_buffers[0];
  hamming_distance<<<tex_height_, num_data_per_block, num_dim_>>>(keys, values, query_device, tex_, tex_height_, num_dim_, num_data_per_block);

  // Allocate temporary storage
  size_t temp_storage_bytes = 0;
  void *d_temp_storage = nullptr;
  CubDebugReturn(DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, num_data_), indexes);
  CubDebugReturn(allocator_.DeviceAllocate(&d_temp_storage, temp_storage_bytes), indexes);

  // Real sort
  d_keys.selector = d_values.selector = 0;
  CubDebugReturn(DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, num_data_), indexes);
  hipDeviceSynchronize();

  // copy to host
  indexes.resize(min(top_k, num_data_));
  CubDebugReturn(hipMemcpy(indexes.data(), d_values.Current(), sizeof(uint32_t) * indexes.size(), hipMemcpyDeviceToHost), indexes);

  return indexes;
}
