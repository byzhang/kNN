#include "kNN_impl.h"
#include "distance.h"

#include <glog/logging.h>

using namespace std;

kNN::impl::~impl() {
  if (tex_) {
    hipDestroyTextureObject(tex_);
  }
  if (data_) {
    hipFreeArray(data_);
  }
}

kNN::impl::impl(const std::vector<uint32_t>& data, uint32_t num_data, uint32_t num_dim)
  : num_data_(num_data), num_dim_(num_dim)
  , tex_height_(num_data / num_data_per_block + ((num_data % num_data_per_block)? 1: 0)) {
  CHECK(data.size() == num_data * num_dim) << "size mismatch:"
      << "data = " << data.size()
      << ", num_data * num_dim = " << num_data * num_dim;

  // allocate array
  auto channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);
  auto error = hipMallocArray(&data_, &channelDesc, num_dim * num_data_per_block, tex_height_);
  CHECK(error == hipSuccess) << "error " << error << " when allocating data_ "
                              << "num_dim:" << num_dim_ << "; num_data:" << num_data_ << "; num_data_per_block:" << num_data_per_block
                              << "; tex_height:" << tex_height_;

  // memcpy array
  error = hipMemcpyToArray(data_, 0, 0, data.data(), num_data_ * num_dim_ * sizeof(uint32_t), hipMemcpyHostToDevice);
  CHECK(error == hipSuccess) << "error " << error << " when copying to data_"
                              << "num_dim:" << num_dim_ << "; num_data:" << num_data_ << "; num_data_per_block:" << num_data_per_block
                              << "; tex_height:" << tex_height_;

  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = data_;

  // Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  // Create texture object
  error = hipCreateTextureObject(&tex_, &resDesc, &texDesc, NULL);
  CHECK(error == hipSuccess) << "error " << error << " when copying to data_"
                              << "num_dim:" << num_dim_ << "; num_data:" << num_data_ << "; num_data_per_block:" << num_data_per_block
                              << "; tex_height:" << tex_height_;
}
